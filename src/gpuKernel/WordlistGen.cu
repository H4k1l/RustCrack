// Copyright 2025 Hakil
// Licensed under the Apache License, Version 2.0

// -- code -- //
#include <sstream>
#include <iomanip>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <cstring>

extern "C" __global__ void genWord(char* chars, int charsLen, unsigned long long* range, int n, char* output){
    
    int i =  blockIdx.x * blockDim.x + threadIdx.x; // calculate the ID of the single thread
    
    if (i < n) {    
        int outpOffset = 0;
        for(int prevthread = 0; prevthread < i; prevthread++) { // calculating the offset of the words with the thread ID
            int prevnumstrings = range[(prevthread*3) + 2] - range[(prevthread*3) + 1] + 1;
            int prevstringlen = range[prevthread*3];
            outpOffset += prevnumstrings * (prevstringlen + 1);
        }

        for(int j = range[(i*3)+1]; j <= range[(i*3)+2]; j++) { // iterate for every seed
            int currentLength = range[i*3];
            int temp = j;

            for(int h = 0; h < currentLength; h++) {// creating the word with the seed

                output[outpOffset + h] = chars[temp % charsLen];
                temp /= charsLen;
            
            }
            outpOffset += currentLength + 1;
        }
    }
}
